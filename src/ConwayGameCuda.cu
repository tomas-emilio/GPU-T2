#include "hip/hip_runtime.h"
#include "ConwayGame.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void conwayKernel(int* inputGrid, int* outputGrid, int rows, int cols){
    int idx = blockIdx.x * blockDim.x +threadIdx.x;
    int idy = blockIdx.y * blockDim.y +threadIdx.y;

    if(idx>=cols || idy >= rows) return;

    int celdaId= idy * cols + idx;
    int vecinosVivos = 0;

    for(int dy=-1; dy<=1; dy++){
        for(int dx=-1; dx<=1; dx++){
            if(dx==0 && dy==0) continue;
            int ny = idy + dy;
            int nx = idx + dx;
            if(ny>=0 && ny<rows && nx>=0 && nx<cols){
                vecinosVivos += inputGrid[ny*cols +nx];
            }
        }
    }

    int celdaActual = inputGrid[celdaId];
    outputGrid[celdaId] = (celdaActual == 1) ? 
        ((vecinosVivos == 2 || vecinosVivos == 3) ? 1 : 0) :
        ((vecinosVivos == 3) ? 1 : 0);
}

__global__ void conwayKernelConIfs(int* inputGrid, int* outputGrid, int rows, int cols){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx >= cols || idy >= rows) return;
    
    int celdaId = idy * cols + idx;
    int vecinosVivos = 0;

    if (idy > 0 && idx > 0 && inputGrid[(idy-1) * cols + (idx-1)] == 1) vecinosVivos++;
    if (idy > 0 && inputGrid[(idy-1) * cols + idx] == 1) vecinosVivos++;
    if (idy > 0 && idx < cols-1 && inputGrid[(idy-1) * cols + (idx+1)] == 1) vecinosVivos++;
    if (idx > 0 && inputGrid[idy * cols + (idx-1)] == 1) vecinosVivos++;
    if (idx < cols-1 && inputGrid[idy * cols + (idx+1)] == 1) vecinosVivos++;
    if (idy < rows-1 && idx > 0 && inputGrid[(idy+1) * cols + (idx-1)] == 1) vecinosVivos++;
    if (idy < rows-1 && inputGrid[(idy+1) * cols + idx] == 1) vecinosVivos++;
    if (idy < rows-1 && idx < cols-1 && inputGrid[(idy+1) * cols + (idx+1)] == 1) vecinosVivos++;
    
    int celdaActual = inputGrid[celdaId];
    outputGrid[celdaId] = (celdaActual == 1) ? 
        ((vecinosVivos == 2 || vecinosVivos == 3) ? 1 : 0) :
        ((vecinosVivos == 3) ? 1 : 0);
}

class ConwayGameCudaImpl : public ConwayGameCuda {
private:
    int* d_grid;
    int* d_nextGrid;
    size_t gridSize;
    bool useIfs;
    int blockSize;
    
public:
    ConwayGameCudaImpl(int rows, int cols, bool useIfs, int blockSize) 
        : ConwayGameCuda(rows, cols), useIfs(useIfs), blockSize(blockSize) {
        this->rows = rows;
        this->cols = cols;
        this->grid.resize(rows * cols, 0);
        this->nextGrid.resize(rows * cols, 0);
        
        gridSize = rows * cols * sizeof(int);
        hipMalloc(&d_grid, gridSize);
        hipMalloc(&d_nextGrid, gridSize);
    }
    
    ~ConwayGameCudaImpl() {
        hipFree(d_grid);
        hipFree(d_nextGrid);
    }
    
    void update() override {
        hipMemcpy(d_grid, grid.data(), gridSize, hipMemcpyHostToDevice);
        
        dim3 blockDim(blockSize, blockSize);
        dim3 gridDim((cols + blockSize - 1) / blockSize, (rows + blockSize - 1) / blockSize);
        
        if (useIfs) {
            conwayKernelConIfs<<<gridDim, blockDim>>>(d_grid, d_nextGrid, rows, cols);
        } else {
            conwayKernel<<<gridDim, blockDim>>>(d_grid, d_nextGrid, rows, cols);
        }
        
        hipDeviceSynchronize();
        hipMemcpy(nextGrid.data(), d_nextGrid, gridSize, hipMemcpyDeviceToHost);
        std::swap(grid, nextGrid);
    }
};

ConwayGameCuda* createConwayGameCuda(int rows, int cols, bool useIfs, int blockSize) {
    return new ConwayGameCudaImpl(rows, cols, useIfs, blockSize);
}

void ConwayGameCuda::update() {
    throw std::runtime_error("Use factory function");
}